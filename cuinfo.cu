// Chasanis Evangelos cs05058
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h" 


int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major){
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        case 7: // Volta and Turing
            if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        case 8: // Ampere
            if (devProp.minor == 0) cores = mp * 64;
            else if (devProp.minor == 6) cores = mp * 128;
            else if (devProp.minor == 9) cores = mp * 128; // ada lovelace
            else printf("Unknown device type\n");
            break;
        case 9: // Hopper
            if (devProp.minor == 0) cores = mp * 128;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n"); 
            break;
    }
    return cores;
}

/* 
 * Retrieves and prints information for every installed NVIDIA
 * GPU device
 */
void cuinfo_print_devinfo() {
    int num_devs, i;
    hipDeviceProp_t dev_prop;

    hipGetDeviceCount(&num_devs);
    if (num_devs == 0) {
        printf("No CUDA devices found.\n");
        return;
    }

    printf("Found %d CUDA device(s):\n", num_devs);
    for (i = 0; i < num_devs; i++) {
        hipGetDeviceProperties(&dev_prop, i);

        printf("Device Name: %s\n", dev_prop.name);
        printf("CUDA Compute Capability: %d.%d\n", dev_prop.major, dev_prop.minor);
        printf("CUDA Driver/Runtime Version: %d.%d\n", CUDART_VERSION / 1000, (CUDART_VERSION % 100) / 10);
        printf("Number of SMs: %d\n", dev_prop.multiProcessorCount);
        printf("Total Global Memory: %lu bytes\n", (unsigned long)dev_prop.totalGlobalMem);
        printf("Total Constant Memory: %lu bytes\n", (unsigned long)dev_prop.totalConstMem);
        printf("Shared Memory Per Block: %lu bytes\n", (unsigned long)dev_prop.sharedMemPerBlock);

        printf("Total CUDA Cores: %d\n", getSPcores(dev_prop));
    }
}

int main() {
    cuinfo_print_devinfo();
    return 0;
}
